#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void func()
{
    printf("Hello world!\n");
}

void kernel()
{
    func<<<1, 1>>>();
    hipDeviceSynchronize();
}