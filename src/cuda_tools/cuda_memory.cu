#include "cuda_memory.cuh"

void cudaXMemcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind)
{
    cuda_safe_call(hipMemcpy(dst, src, count, kind));
}

void cudaXMemcpyAsync(void* dst, const void* src, size_t count, hipMemcpyKind kind, hipStream_t stream)
{
    cuda_safe_call(hipMemcpyAsync(dst, src, count, kind, stream));
}

void cudaXMemset(void* devPtr, int  value, size_t count)
{
    cuda_safe_call(hipMemset(devPtr, value, count));
}

void cudaXFree(void* devPtr)
{
    cuda_safe_call(hipFree(devPtr));
}